#include "hip/hip_runtime.h"
#include <optix.h>
#include <cuda/helpers.h>

#include "Programs\ray.cu"

extern "C" __global__ void __closesthit__radiance()
{
    PhongHitGroupData* hit_data = reinterpret_cast<PhongHitGroupData*>(optixGetSbtDataPointer());
    const int primID = optixGetPrimitiveIndex();

    const ushort3 index =
        hit_data->indices == NULL
        ? ushort3{
            (unsigned short)(3 * primID + 0),
            (unsigned short)(3 * primID + 1),
            (unsigned short)(3 * primID + 2) }
    : hit_data->indices[primID];
    const float3 A = hit_data->vertices[index.x];
    const float3 B = hit_data->vertices[index.y];
    const float3 C = hit_data->vertices[index.z];

    const float3 objectN = cross(B - A, C - A);
    const float3 worldN = normalize(optixTransformNormalFromObjectToWorldSpace(objectN));

    const float t = optixGetRayTmax();
    float3 worldHitPosition = optixGetWorldRayOrigin() + t * optixGetWorldRayDirection();

    const auto lights_position = reinterpret_cast<float3*>(hit_data->lights_position);
    const auto lights_color = reinterpret_cast<float3*>(hit_data->lights_color);
    float3 cameraDir = -optixGetWorldRayDirection();
    float3 color = hit_data->ambient_color;

    for (int i = 0; i < hit_data->point_lights_count; i++) {
        float3 lightDir = normalize(lights_position[i] - worldHitPosition);

        float cosTheta = fmax(dot(worldN, lightDir), 0.0f);
        float3 diffuse = cosTheta * lights_color[i];
        color = 1 - (1 - color) * (1 - diffuse);

        float3 reflectDir = reflect(-lightDir, worldN);
        float intensity = pow(max(dot(cameraDir, reflectDir), 0.0f), hit_data->material_shininess);
        float3 specular = intensity * lights_color[i] * hit_data->material_color_specular;
        color = 1 - (1 - color) * (1 - specular);
    }

    float4 material_color4 = hit_data->material_color;
    float3 material_color3 = float3{ 
        material_color4.x, 
        material_color4.y, 
        material_color4.z 
    };
    setPayload(color * material_color3);
}

